#include <hip/hip_runtime.h>

#include "raytrace.cuh"
#include "hittable_list.cuh"
#include "sphere.cuh"
#include "camera.cuh"
#include "material.cuh"

int main() {
    hittable_list world;

    sphere* spheres = (sphere*)malloc(sizeof(sphere) * 2);
    spheres[0] = sphere(point3(0, 0, -1), 0.5, LAMBERTIAN);
    spheres[1] = sphere(point3(0, -100.5, -1), 100, LAMBERTIAN);

    camera cam;

    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_width  = 400;

    cam.render(spheres);
    return 0;
}